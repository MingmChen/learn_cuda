#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <unistd.h>

using namespace std;


__global__ void add(int a, int b, int *c)   //kernel函数，在gpu上运行。
{
    *c = a + b;
}
 
int main()
{
    while(1)
    {
        int c;
        int *dev_c;

        // 分配gpu的内存，第一个参数指向新分配内存的地址，第二个参数是分配内存的大小
        hipMalloc((void**)&dev_c, sizeof(int));    
        hipMemset(dev_c, 0, sizeof(int))

        // 调用kernel函数，<<<1,1>>>指gpu启动1个线程块，每个线程块中有1个线程
        add<<<1,1>>>(2, 7, dev_c);  

        // 将gpu上的数据复制到主机上，
        // 即从dev_c指向的存储区域中将sizeof(int)个字节复制到&c指向的存储区域
        hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost); 

        cout << "2 + 7 = " << c << endl;

        //释放hipMalloc分配的内存
        hipFree(dev_c);

        sleep(2);
        
    }
    return 0;
}