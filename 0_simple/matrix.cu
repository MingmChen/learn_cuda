#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <unistd.h>

using namespace std;

/**
 * 矩阵相乘,假设是方阵
 * @param Md    [矩阵]
 * @param Nd    [矩阵]
 * @param Pd    [结果矩阵]
 * @param width [矩阵宽度]
 */
__global__ void matrix_mulit_kernel(float *Md, float *Nd, float *Pd, int width)
{
    // 二维的线程块索引
    // main 调用的时候，block的大小是16x16
    // 即 threadIdx([0,15], [0,15]), 运行之后，将会有16个线程块同时执行这段代码
    // 第一个线程 threadIsx.x = 0, threadIsx.y = 0
    // 也即 for循环的运算的是 矩阵 Md 的第0行和 矩阵 N 的第0列相乘的结果。
    // 
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    float pValue = 0;

    for (int i = 0; i < width; i++)
    {   
        // ty 在第 0 个线程中所以取的是矩阵第一行的值
        float m = Md[ty*width + i];

        // tx 在第 0 个线程中
        // i= 0 时，取第一列第一个元素
        // i= 1 时，取第二列第一个元素
        // ……
        float n = Nd[i*width + tx]; 

        // M 的行乘以 N 的列 并且相加
        pValue += m*n; 
    }

    Pd[ty*width+tx] = pValue;
}

int main(int argc, char const *argv[]) 
{
    int width = 16;

    dim3 dimBlock(width, width);
    dim3 dimGrid(1, 1);

    matrix_mulit_kernel<<<dimGrid, dimBlock>>>();

    return 0;
}

