

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>


// __global__ 修饰符，将告诉编译器，函数在设备(GPU)上运行而不是在主机(CPU)上运行
__global__ void kernel(void)
{
    printf("Hello world!\n");
}

int main(void)
{
    while(1)
    {
        kernel<<<1,1>>>();

        sleep(1);
    }   

    return 0;
}

// compile
// nvcc hello.cu -o hello