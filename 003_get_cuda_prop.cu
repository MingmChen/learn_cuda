/**
 * 获取GPU属性
 */

#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

int main(int argc, char const *argv[]) 
{
    hipDeviceProp_t prop;

    int count;

    // 获取有所少快GPU设备
    hipGetDeviceCount(&count);

    for(unsigned i = 0; i < count; ++i) 
    {   
        // 获取GPU属性信息
        hipGetDeviceProperties(&prop, i);
        cout << "name:      " << prop.name << endl;
        cout << "totalGlobalMem:      " << prop.totalGlobalMem << endl;
        cout << "sharedMemPerBlock: " << prop.sharedMemPerBlock << endl;
        cout << "regsPerBlock: " << prop.regsPerBlock << endl;
        cout << "warpSize: " << prop.warpSize << endl;
        cout << "memPitch: " << prop.memPitch << endl;
        cout << "canMapHostMemory: " << prop.canMapHostMemory << endl;
        cout << "pciDeviceID: " << prop.pciDeviceID << endl;
        cout << "tccDriver: " << prop.tccDriver << endl;
        cout << "----------------------------------------------------"<< endl;
    }

    // 设置当前使用那块GPU
    hipSetDevice(count);

    return 0;
}