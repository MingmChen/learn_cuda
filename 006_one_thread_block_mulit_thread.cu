/**
 * 并行计算
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include<sys/time.h>

using namespace std;

#define N (200000)

void add_cpu(int *a, int *b, int *c)
{
    int tid = 0;

    while(tid < N) 
    {
        c[tid] = a[tid] + b[tid];
        tid += 1;
        /* code */
    }
}

__global__ void add(int *a, int *b, int *c)
{
    int tid = blockIdx.x;

    while(tid < N) 
    {
        c[tid] = a[tid] + b[tid];
        tid += 1;
        /* code */
    }
}

// CPU 求和
int main_cpu() 
{
    int a[N], b[N], c[N];

    struct timeval tv1, tv2;

    
    for (int i = 0; i < N; i++)
    {
        a[i] = -i;
        b[i] = i*i;
    }

    gettimeofday(&tv1, NULL);
    add_cpu(a, b, c);
    gettimeofday(&tv2, NULL);
    float time = (1000000 * (tv2.tv_sec - tv1.tv_sec) + tv2.tv_usec- tv1.tv_usec)/1000.0;
    cout << "time cpu： " << time << "ms, num : " << c[N-1] << endl;
    
    return 0;
}

// GPU 求和
int main(int argc, char const *argv[]) 
{   
    int a[N], b[N], c[N];

    int *dev_a, *dev_b, *dev_c;

    struct timeval tv1, tv2;

    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    // 在CPU上为数组 a/b赋值
    // 这里在CPU就给输出数据赋初值，并没有特殊原因。事实上，如果在GPU上对数组赋值，这个步骤执行的会更快。
    // 但是这段代码的目的是说明如何在显卡上实现两个矢量的加法运算，因此我们仅仅将计算部分放在显卡上实现，
    // 输入则在CPU上进行。
    for(unsigned i = 0; i < N; ++i) 
    {
        a[i] = -i;
        b[i] = i*i;
    }

    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, N * sizeof(int), hipMemcpyHostToDevice);

    gettimeofday(&tv1, NULL);

    // 调用kernel函数，<<<1,1>>>指gpu启动1个线程块，每个线程块中有1个线程
    // <<<256,1>>>指gpu启动256个线程块，每个线程块中有1个线程, 如果是这样，就会有一个问题：
    // 既然GPU将运行核函数的N个副本，那如何在代码中知道当前正在运行的是哪一个线程块？
    // 这个问题可以在代码中找出答案：
    // int tid = blockIdx.x
    // 乍一看，将一个没有定义的变量赋值给了变量tid,但是 blockIdx 是一个内置变量，在cuda运行是中已经定义了。
    // 这个变量把包含的值就是当前执行设备代码的的线程块索引。
    // 
    // 问题又来了，为什么不是写 int  tid = blockIdx呢？ 事实上，这是因为cuda支持二维的线程块数组，对于二维空间的计算问题，
    // 例如矩阵数学运算或者图像处理，使用二维索引往往回答来很大的便利，因为他可以避免将线性索引转换为矩形索引。
    add<<<1, 65535>>>(dev_a, dev_b, dev_c);

    gettimeofday(&tv2, NULL);
    float time = (1000000 * (tv2.tv_sec - tv1.tv_sec) + tv2.tv_usec- tv1.tv_usec)/1000.0;
    cout << "time gpu： " << time << "ms";


    hipMemcpy(c, dev_c,  N * sizeof(int), hipMemcpyDeviceToHost);
    cout << ", num : " << c[N-1] << endl;

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    main_cpu();

    /* code */
    return 0;
}

// time gpu： 0.048ms
// time cpu： 1.248