#include "hip/hip_runtime.h"

//1）CUDA和OPENCV联系起来；（test1.cu)
// 需要gui环境

#include <iostream>
#include <unistd.h>
#include "opencv2/core/core.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

using namespace std;
using namespace cv;
#define N 250
//test1的kernel
__global__ void test1kernel(int *t)
{
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x+y*gridDim.x; 
    t[offset] =255-t[offset];
 
}
int main(void)
{    
    //step0.数据和内存初始化
    Mat src = imread("opencv-logo.png",0);
    resize(src,src,Size(N,N));
    int *dev_t;
    int t[N*N];
    Mat dst = Mat(N,N,CV_8UC3);
    for (int i=0;i<N*N;i++)
    {
        t[i]  =(int)src.at<char>(i/N,i%N);
    }
    checkCudaErrors(hipMalloc((void **)&dev_t, sizeof(int)*N*N));
    //step1.由cpu向gpu中导入数据
    checkCudaErrors(hipMemcpy(dev_t, t,sizeof(int)*N*N, hipMemcpyHostToDevice));
    //step2.gpu运算
    dim3 grid(N,N);
    test1kernel<<<grid,1>>>(dev_t);
    //step3.由gpu向cpu中传输数据
    checkCudaErrors(hipMemcpy(t, dev_t,sizeof(int)*N*N, hipMemcpyDeviceToHost));
    //step4.显示结果
    for (int i=0;i<N;i++)
    {
        for (int j=0;j<N;j++)
        {
             int offset = i*N+j;
             for (int c=0;c<3;c++)
             {
                 dst.at<Vec3b>(i,j)[c] =t[offset];
             }
        }
    }
    //step5，释放资源
    checkCudaErrors(hipFree(dev_t));
    // imshow("dst",dst);
    waitKey();
    return 0;
}