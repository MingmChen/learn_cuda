#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include <stdio.h>

#include "nv12_to_rgb.h"

__global__ void dev_nv12_to_rgb(unsigned char *pYdata, unsigned char *pUVdata, int stepY, int stepUV, unsigned char *pImgData, int width, int height, int channels)
{
    const int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    const int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    // printf("x-->blockIdx.x, blockDim.x threadIdx.x [%d %d %d] \n", blockIdx.x, blockDim.x, threadIdx.x);
    // printf("y-->blockIdx.y, blockDim.y threadIdx.y [%d %d %d] \n", blockIdx.y, blockDim.y, threadIdx.y);

    if (tidx < width && tidy < height)
    {
        int indexY, indexU, indexV;
        unsigned char Y, U, V;
        indexY = tidy * stepY + tidx;    
        Y = pYdata[indexY];

        if (tidx % 2 == 0)
        {
            indexU = tidy / 2 * stepUV + tidx;
            indexV = tidy / 2 * stepUV + tidx + 1;
            U = pUVdata[indexU];
            V = pUVdata[indexV];
        }
        else if (tidx % 2 == 1)
        {
            indexV = tidy / 2 * stepUV + tidx;
            indexU = tidy / 2 * stepUV + tidx - 1;
            U = pUVdata[indexU];
            V = pUVdata[indexV];
        }

        pImgData[(tidy*width + tidx) * channels + 2] = uchar (Y + 1.402 * (V - 128));
        pImgData[(tidy*width + tidx) * channels + 1] = uchar (Y - 0.34413 * (U - 128) - 0.71414*(V - 128));
        pImgData[(tidy*width + tidx) * channels + 0] = uchar (Y + 1.772*(U - 128));
    }
}

bool nv12_to_rgb(unsigned char *src, unsigned char *dst, int src_width, int src_height, int dst_pitch)
{
    dim3 block(32,8);

    printf("block.x, block.y [%d %d] \n", block.x, block.y);

    int gridx = (src_width + block.x )/(block.x);
    int gridy = (src_height + block.y )/(block.y);

    printf("gridx, gridy [%d %d]\n", gridx, gridy);

    dim3 grid(gridx, gridy);

    unsigned char *Y, *UV;
    unsigned char *y  = src;
    unsigned char *uv = src+(src_width * src_height);
    int y_size = src_width * src_height * sizeof(unsigned char);
    int uv_size = y_size / 2;

    hipMalloc((void**)&Y,  y_size);
    hipMalloc((void**)&UV, uv_size);
    hipMemcpy(Y, y, y_size, hipMemcpyHostToDevice);
    hipMemcpy(UV, uv, uv_size, hipMemcpyHostToDevice);

    unsigned char *d_dst;
    int src_mem_size = sizeof(unsigned char ) * src_width * src_height * 3;
    hipMalloc((void**)&d_dst, src_mem_size);
    hipMemcpy(d_dst, d_dst, src_mem_size, hipMemcpyHostToDevice);

    dev_nv12_to_rgb<<<grid,block>>>(Y, UV, src_width, src_width, d_dst, src_width, src_height, 3);

    hipMemcpy(dst, d_dst, src_mem_size, hipMemcpyDeviceToHost);

    hipFree(d_dst);
    hipFree(Y);
    hipFree(UV);

    return true;
} 