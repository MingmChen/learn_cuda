#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <iostream>

#include "yuv_resize.h"

// __global__ void device_yuv_resize(unsigned char *d_y, unsigned char *d_uv, int step_y, 
//     int step_uv, unsigned char *d_dst, int dst_width, int dst_height)
// {
//     const int tidx = blockIdx.x * blockDim.x + threadIdx.x;
//     const int tidy = blockIdx.y * blockDim.y + threadIdx.y;

//     if (tidx < dst_width && tidy < dst_height)
//     {
//         int index_y, index_u, index_v;
//         unsigned char Y00, Y01, Y10, Y11, U00, V01, U10, V11, Y , U, V;
//         index_y = tidy * step_y + tidx;    

//         // if (tidx % 2 == 0 && tidx < 3)
//         // {
//         //     printf("%d %d %d %d\n", tidy*step_y + tidx, tidy*step_y + tidx + 1, (tidy+1)*step_y + tidx, (tidy+1)*step_y + tidx + 1);
//         // }

//         if (tidx % 2 == 0)
//         {
//             Y00 = d_y[tidy*step_y + tidx];
//             Y01 = d_y[tidy*step_y + tidx + 1];
//             Y10 = d_y[(tidy+1)*step_y + tidx];
//             Y11 = d_y[(tidy+1)*step_y + tidx + 1];

//             Y = (Y00 + Y01 + Y10 + Y11) / 4;
//         }

//         if (tidx % 2 == 0)
//         {
//             U00 = d_uv[tidy / 2 * step_uv + tidx];
//             V01 = d_uv[tidy / 2 * step_uv + tidx + 1];

//             U10 = d_uv[(tidy / 2 + 1) * step_uv + tidx];
//             V11 = d_uv[(tidy / 2 + 1) * step_uv + tidx + 1];

//             U = (U00 + U10) / 2;
//             V = (V01 + V11) / 2;
//         }

//         int y_size = dst_width * dst_height;

//         d_dst[tidy*dst_width + tidx] = uchar(Y);
//         d_dst[y_size + tidy/2*dst_width + tidx] = uchar (U);
//         d_dst[y_size + tidy/2*dst_width + tidx] = uchar (V);
//     }
// }

// bool host_yuv_resize(unsigned char *src, int src_width, int src_height, unsigned char *dst, int dst_width, int dst_height)
// {
//     dim3 block(32,8);
//     // int gridx = (src_width + block.x )/(block.x);
//     // int gridy = (src_height + block.y )/(block.y);

//     int gridx = (dst_width + block.x )/(block.x);
//     int gridy = (dst_height + block.y )/(block.y);

//     dim3 grid(gridx, gridy);

//     unsigned char *y  = src;
//     unsigned char *uv = src+(src_width * src_height);
//     int y_size = src_width * src_height * sizeof(unsigned char);
//     int uv_size = y_size / 2;

//     unsigned char *d_y, *d_uv;
//     hipMalloc((void**)&d_y,  y_size);
//     hipMalloc((void**)&d_uv, uv_size);
//     hipMemcpy(d_y, y, y_size, hipMemcpyHostToDevice);
//     hipMemcpy(d_uv, uv, uv_size, hipMemcpyHostToDevice);

//     unsigned char *d_dst;
//     int yuv_size = sizeof(unsigned char ) * dst_width * dst_height * 3 / 2;
//     hipMalloc((void**)&d_dst, yuv_size);
//     hipMemcpy(d_dst, d_dst, yuv_size, hipMemcpyHostToDevice);

//     device_yuv_resize<<<grid,block>>>(d_y, d_uv, src_width, src_width, src_width, src_height, d_dst, dst_width, dst_height);

//     hipMemcpy(dst, d_dst, yuv_size, hipMemcpyDeviceToHost);

//     for (int i = 0; i < 1000; i++)
//     {
//         printf("%d ", dst[i]);
//     }

//     hipFree(d_dst);
//     hipFree(d_y);
//     hipFree(d_uv);

//     return true;
// } 

/*
__global__ void device_yuv_resize(unsigned char *d_y, unsigned char *d_uv, int step_y, int step_uv, 
    int src_width, int src_height, unsigned char *d_dst, int dst_width, int dst_height)
{
    const int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    const int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    unsigned char Y00, Y01, Y10, Y11, U00, U01, U10, U11, V00, V01, V10, V11, Y , U, V;

    if (tidx < src_width && tidy < src_height)
    {
        if (tidx % 2 == 0 && tidy % 2 == 0)
        {
            Y00 = d_y[tidy*step_y + tidx];
            Y01 = d_y[tidy*step_y + tidx + 1];
            Y10 = d_y[(tidy+1)*step_y + tidx];
            Y11 = d_y[(tidy+1)*step_y + tidx + 1];

            // printf("%d %d  %d %d\n", tidy*step_y + tidx, tidy*step_y + tidx + 1, (tidy+1)*step_y + tidx, (tidy+1)*step_y + tidx + 1);

            Y = (unsigned char)(Y00 + Y01 + Y10 + Y11) / 4;

            d_dst[tidy/2*dst_width + tidx/2] = uchar(Y);

            // if ((tidy/2*dst_width + tidx/2)  > 522200)
            // {
            //     printf("%6d ", tidy/2*dst_width + tidx/2);
            // }
        }

        // if (tidy < (src_height / 2))
        // {
            if (tidx % 4 == 0 && tidy % 2 == 0)
            {
                U00 = d_uv[tidy * step_uv + tidx];
                U01 = d_uv[tidy * step_uv + tidx + 2];
                U10 = d_uv[(tidy + 1) * step_uv + tidx];
                U11 = d_uv[(tidy + 1) * step_uv + tidx + 2];

                V00 = d_uv[tidy * step_uv + tidx + 1];
                V01 = d_uv[tidy * step_uv + tidx + 3];
                V10 = d_uv[(tidy + 1) * step_uv + tidx + 1];
                V11 = d_uv[(tidy + 1) * step_uv + tidx + 3];

                U = (U00 + U01 + U10 + U11) / 4;
                V = (V00 + V01 + V10 + V11) / 4; 

                int y_size = dst_width * dst_height;
                if ((tidy/2*dst_width + tidx/4) % 2 == 0)
                {
                    d_dst[y_size + tidy/2*dst_width + tidx/4] = uchar (U);
                    d_dst[y_size + tidy/2*dst_width + tidx/4 + 1] = uchar (V);

                    if ((tidy/2*dst_width + tidx/4) > 261100)
                    {
                        printf("%6d ", tidy/2*dst_width + tidx/4);
                    }
                    
                }
            }
        // }
    }    
}
*/

/**
 * 基于GPU 做 YUV (NV12)数据压缩，压缩至原来的一般 1920x1080 --> 960x540
 * @param d_y        [Y 数据]
 * @param d_uv       [UV 数据]
 * @param step_y     [Y 数据的宽度]
 * @param step_uv    [uv 数据的宽度]
 * @param src_width  [原始数据宽]
 * @param src_height [原始数据高]
 * @param d_dst      [压缩后的yuv 输出参数]
 * @param dst_width  [压缩后的宽]
 * @param dst_height [压缩后的高]
 */
__global__ void device_yuv_resize(unsigned char *d_y, unsigned char *d_uv, int step_y, int step_uv, 
    int src_width, int src_height, unsigned char *d_dst, int dst_width, int dst_height)
{
    const int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    const int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    unsigned char Y00, Y01, Y10, Y11, U00, U01, U10, U11, V00, V01, V10, V11, Y , U, V;

    if (tidx < src_width && tidy < src_height)
    {
        if (tidx % 2 == 0 && tidy % 2 == 0)
        {
            Y00 = d_y[tidy*step_y + tidx];
            Y01 = d_y[tidy*step_y + tidx + 1];
            Y10 = d_y[(tidy+1)*step_y + tidx];
            Y11 = d_y[(tidy+1)*step_y + tidx + 1];

            Y = uchar((Y00 + Y01 + Y10 + Y11) / 4);

            d_dst[tidy/2*dst_width + tidx/2] = uchar(Y);
        }

        if (tidx % 4 == 0 && tidy % 4 == 0)
        {
            U00 = d_uv[tidy/2 * step_uv + tidx];
            U01 = d_uv[tidy/2 * step_uv + tidx + 2];
            U10 = d_uv[(tidy/2 + 1) * step_uv + tidx];
            U11 = d_uv[(tidy/2 + 1) * step_uv + tidx + 2];

            V00 = d_uv[tidy/2 * step_uv + tidx + 1];
            V01 = d_uv[tidy/2 * step_uv + tidx + 3];
            V10 = d_uv[(tidy/2 + 1) * step_uv + tidx + 1];
            V11 = d_uv[(tidy/2 + 1) * step_uv + tidx + 3];

            U = (U00 + U01 + U10 + U11) / 4;
            V = (V00 + V01 + V10 + V11) / 4; 

            int y_size = dst_width * dst_height;
            if ((tidy/4*dst_width + tidx/2) % 2 == 0)
            {
                d_dst[y_size + tidy/4*dst_width + tidx/2] = uchar (U);
                d_dst[y_size + tidy/4*dst_width + tidx/2 + 1] = uchar (V);
            }
        }
    }    
}

/**
 * Host 接口 压缩YUV数据
 * @param  src        [YUV 原始数据]
 * @param  src_width  [原始宽]
 * @param  src_height [原始高]
 * @param  dst        [压缩后的YUV数据， 输出参数]
 * @param  dst_width  [压缩后宽]
 * @param  dst_height [研所后高]
 * @return            [description]
 */
bool host_yuv_resize(unsigned char *src, int src_width, int src_height, unsigned char *dst, int dst_width, int dst_height)
{
    dim3 block(32,8);
    int gridx = (src_width + block.x )/(block.x);
    int gridy = (src_height + block.y )/(block.y);
    dim3 grid(gridx, gridy);

    unsigned char *y  = src;
    unsigned char *uv = src+(src_width * src_height);
    int y_size = src_width * src_height * sizeof(unsigned char);
    int uv_size = y_size / 2;

    unsigned char *d_y, *d_uv;
    hipMalloc((void**)&d_y,  y_size);
    hipMalloc((void**)&d_uv, uv_size);
    hipMemcpy(d_y, y, y_size, hipMemcpyHostToDevice);
    hipMemcpy(d_uv, uv, uv_size, hipMemcpyHostToDevice);

    unsigned char *d_dst;
    int yuv_size = sizeof(unsigned char ) * dst_width * dst_height * 3 / 2;
    hipMalloc((void**)&d_dst, yuv_size);
    hipMemcpy(d_dst, d_dst, yuv_size, hipMemcpyHostToDevice);

    device_yuv_resize<<<grid,block>>>(d_y, d_uv, src_width, src_width, src_width, src_height, d_dst, dst_width, dst_height);

    hipMemcpy(dst, d_dst, yuv_size, hipMemcpyDeviceToHost);

    hipFree(d_dst);
    hipFree(d_y);
    hipFree(d_uv);

    return true;
} 

/**
 * GPU 端核函数：NV12 数据转RGB
 * @param pYdata   [Y数据]
 * @param pUVdata  [UV数据]
 * @param stepY    [Y数据的宽度]
 * @param stepUV   [UV数据的宽度]
 * @param pImgData [输出参数 RGB数据]
 * @param width    [输出宽]
 * @param height   [输出高]
 * @param channels [RGB 通道数]
 */
__global__ void dev_nv12_to_rgb(unsigned char *pYdata, unsigned char *pUVdata, int stepY, 
    int stepUV, unsigned char *pImgData, int width, int height, int channels)
{
    const int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    const int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    if (tidx < width && tidy < height)
    {
        int index_y, index_u, index_v;
        unsigned char Y, U, V;
        index_y = tidy * stepY + tidx;    
        Y = pYdata[index_y];

        if (tidx % 2 == 0)
        {
            index_u = tidy / 2 * stepUV + tidx;
            index_v = tidy / 2 * stepUV + tidx + 1;
            U = pUVdata[index_u];
            V = pUVdata[index_v];
        }
        else if (tidx % 2 == 1)
        {
            index_v = tidy / 2 * stepUV + tidx;
            index_u = tidy / 2 * stepUV + tidx - 1;
            U = pUVdata[index_u];
            V = pUVdata[index_v];
        }

        pImgData[(tidy*width + tidx) * channels + 2] = uchar (Y + 1.402 * (V - 128));
        pImgData[(tidy*width + tidx) * channels + 1] = uchar (Y - 0.34413 * (U - 128) - 0.71414*(V - 128));
        pImgData[(tidy*width + tidx) * channels + 0] = uchar (Y + 1.772*(U - 128));
    }
}

/**
 * Host 接口： YUV 转 RGB
 * @param  src        [原始YUV]
 * @param  dst        [转完之后的RGB数据]
 * @param  src_width  [原始宽]
 * @param  src_height [原始高]
 * @param  dst_pitch  [转换后的宽度]
 * @return            [description]
 */
bool nv12_to_rgb(unsigned char *src, unsigned char *dst, int src_width, int src_height, int dst_pitch)
{
    dim3 block(32,8);
    int gridx = (src_width + block.x )/(block.x);
    int gridy = (src_height + block.y )/(block.y);
    dim3 grid(gridx, gridy);

    unsigned char *Y, *UV;
    unsigned char *y  = src;
    unsigned char *uv = src+(src_width * src_height);
    int y_size = src_width * src_height * sizeof(unsigned char);
    int uv_size = y_size / 2;

    hipMalloc((void**)&Y,  y_size);
    hipMalloc((void**)&UV, uv_size);
    hipMemcpy(Y, y, y_size, hipMemcpyHostToDevice);
    hipMemcpy(UV, uv, uv_size, hipMemcpyHostToDevice);

    unsigned char *d_dst;
    int src_mem_size = sizeof(unsigned char ) * src_width * src_height * 3;
    hipMalloc((void**)&d_dst, src_mem_size);
    hipMemcpy(d_dst, d_dst, src_mem_size, hipMemcpyHostToDevice);

    dev_nv12_to_rgb<<<grid,block>>>(Y, UV, src_width, src_width, d_dst, src_width, src_height, 3);

    hipMemcpy(dst, d_dst, src_mem_size, hipMemcpyDeviceToHost);

    hipFree(d_dst);
    hipFree(Y);
    hipFree(UV);

    return true;
} 
